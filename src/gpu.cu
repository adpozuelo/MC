#include "hip/hip_runtime.h"
/*
 * TFM (URV/UOC): Computational Engineering and Mathematics.
 * Serial and parallel (CUDA) general purpose Monte Carlo code for atomistic
 * simulations.
 *
 * GPU functions code file
 *
 * Author: adpozuelo@gmail.com
 * Version: 2.0
 * Date: 2020
 */

 #include <assert.h>
 #include <hiprand.h>
 #include <hiprand/hiprand_kernel.h>
 #include <stdio.h>
 #include <stdlib.h>
 extern "C" {
 #include "../include/io.h"
 #include "mkl_vsl.h"
 }
 
 // GPU (device code) interaction type (id) of a particle (see util.cpp for CPU
 // code).
 __device__ int __getIatype__(int nsp, int *nspps, int sp) {
   int i = 0;
   for (; i < nsp; ++i) {
	 if (sp < nspps[i]) {
	   break;
	 }
   }
   return i;
 }
 
 // GPU (device code) dot/scalar product => distance (power 2) (see util.cpp for
 // CPU code).
 __device__ float __dist2__(float *r, float *runit) {
   for (int i = 0; i < NDIM; ++i) {
	 if (r[i] > 0.5) {
	   r[i] -= 1;
	 }
	 if (r[i] < -0.5) {
	   r[i] += 1;
	 }
   }
   float rd2 = 0.0;
   for (int i = 0; i < NDIM; ++i) {
	 r[i] *= runit[i];
	 r[i] *= r[i];
	 rd2 += r[i];
   }
   return rd2;
 }
 
 // GPU (device code) energy between two particles (see potEnergy.cpp for CPU
 // code).
 __device__ float __fpot__(float r2, int nit, int *keyp, float *al, float *bl,
						   float *cl, float *bl2) {
   float r, r6;
   if (keyp[nit] == 1) {
	 float rr = __fsqrt_rn(r2);
	 float expp = __expf(-bl[nit] * (rr - cl[nit]));
	 r = al[nit] * ((1 - expp) * (1 - expp) - 1.0);
   } else if (keyp[nit] == 2) {
	 r6 = (bl2[nit] / r2) * (bl2[nit] / r2) * (bl2[nit] / r2);
	 r = 4 * al[nit] * r6 * (r6 - 1.0);
   } else {
	 asm("trap;");
   }
   return r;
 }
 
 // GPU chemical potential CUDA kernel.
 __global__ void chpotKernel(float *r, float *chpot, int natoms, int nsp,
							 int *itp, float *runit, float *rc2, float *al,
							 float *bl, float *bl2, float *cl, int *nspps,
							 int *keyp, int sp, unsigned int seed, int chpotit,
							 float kt) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
   int tx = threadIdx.x;                             // block thread index
   hiprandState_t state;                // CuRand state
   hiprand_init(seed, idx, 0, &state);  // CuRand stream initialization
   __shared__ float deltae[NTHREAD];  // energy changes related to the excess
									  // chemical potential (block vector)
   float xyz[NDIM], rdd[NDIM];
   float rd2, eng;
   int iti, itj, nit;
   // if global thread index is a valid chemical potential iteration; thus, every
   // thread inserts one particle
   if (idx < chpotit) {
	 for (int i = 0; i < NDIM; ++i) {
	   // each global thread index generate its own particle's position from
	   // CuRand uniform random number
	   xyz[i] = hiprand_uniform(&state);
	 }
	 // the id for the inserted particle
	 iti = sp;
	 // energy accumulator set to zero
	 eng = 0.0;
	 // for every particle in the configuration
	 for (int n = 0; n < natoms; ++n) {
	   // the id for the configuration particle
	   itj = __getIatype__(nsp, nspps, n);
	   // get interaction potential id between particles
	   nit = itp[iti * nsp + itj];
	   for (int k = 0; k < NDIM; ++k) {
		 // calculate distante between particles (inserted and existing)
		 rdd[k] = xyz[k] - r[n * NDIM + k];
	   }
	   // distance power 2 between particles (inserted and existing)
	   rd2 = __dist2__(rdd, runit);
	   // if distance is less than interaction pontetial cutoff radio
	   if (rd2 < rc2[nit]) {
		 // accumulate energy between particles
		 eng += __fpot__(rd2, nit, keyp, al, bl, cl, bl2);
	   }
	 }
	 // energy changes related to the excess chemical potential
	 deltae[tx] = __expf(-eng / kt);
	 // if global thread index is not a valid chemical potential iteration
   } else {
	 // energy changes related to the excess chemical potential set to zero
	 deltae[tx] = 0.0;
   }
   __syncthreads();  // sync block threads
   // binary reduction energy changes block vectors
   for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
	 // for every thread in the lower middle half of the block
	 if (tx < s) {
	   // sum energy changes of its block index and its block index plus half the
	   // size of the block
	   deltae[tx] += deltae[tx + s];
	 }
	 __syncthreads();  // sync block threads
   }
   // if block thread index is zero (the first thread of the block)
   if (tx == 0) {
	 // atomic (transactional) sum block energy changes to energy accumulator
	 atomicAdd(chpot, deltae[0]);
   }
 }
 
 // GPU delta energy (MoveVolume algorithm) CUDA kernel.
 __global__ void deKernel(int ntest, float *r, float *eng0, float *eng1,
						  int natoms, float *rp, int nsp, int *itp, float *runit,
						  float *rc2, float *al, float *bl, float *bl2,
						  float *cl, int *nspps, int *keyp) {
   // energy before and after particle movement (block vectors)
   __shared__ float e0[NTHREAD], e1[NTHREAD];
   int j = threadIdx.x + blockIdx.x * blockDim.x;  // global thread index
   int tx = threadIdx.x;                           // block thread index
   float rdd[NDIM], rddn[NDIM];
   float rd2;
   int iti, itj, nit;
   // if global thread index is a valid atom/particle
   if (j < natoms) {
	 // if global thread index is the particle to be moved (is the same particle)
	 if (j == ntest) {
	   // set energies to zero
	   e0[tx] = 0.0;
	   e1[tx] = 0.0;
	 }
	 // if global thread index is not the particle to be moved (is not the same
	 // particle)
	 if (j != ntest) {
	   for (int k = 0; k < NDIM; ++k) {
		 // calculate distante between particles (before and after the movement)
		 rdd[k] = r[j * NDIM + k] - r[ntest * NDIM + k];
		 rddn[k] = r[j * NDIM + k] - rp[k];
	   }
	   // get interaction potential ids for the particles
	   iti = __getIatype__(nsp, nspps, ntest);
	   itj = __getIatype__(nsp, nspps, j);
	   // get interaction potential id between particles
	   nit = itp[iti * nsp + itj];
	   // after movement distance power 2 between particles
	   rd2 = __dist2__(rdd, runit);
	   // if distance is less than interaction pontetial cutoff radio
	   if (rd2 < rc2[nit]) {
		 // get energy between particles
		 e0[tx] = __fpot__(rd2, nit, keyp, al, bl, cl, bl2);
		 // if distance is more than interaction potential particles don't
		 // interact
	   } else {
		 e0[tx] = 0.0;  // energy set to zero
	   }
	   // before movement
	   rd2 = __dist2__(rddn, runit);
	   if (rd2 < rc2[nit]) {
		 e1[tx] = __fpot__(rd2, nit, keyp, al, bl, cl, bl2);
	   } else {
		 e1[tx] = 0.0;
	   }
	 }
   } else {  // if global thread index is not a valid atom/particle
	 // set energies to zero
	 e0[tx] = 0.0;
	 e1[tx] = 0.0;
   }
   __syncthreads();  // sync block threads before reduce block energies vectors
   // binary reduction energies block vectors
   for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
	 if (tx < s) {  // for every thread in the lower middle half of the block
	   e0[tx] += e0[tx + s];  // sum energies of its block index and its block
							  // index plus half the size of the block
	   e1[tx] += e1[tx + s];
	 }
	 __syncthreads();  // sync block threads
   }
   // if block thread index is zero (the first thread of the block) atomic
   // (transactional) sum block energy to energy accumulator
   if (tx == 0) {
	 atomicAdd(eng0, e0[0]);  // energy after the particle's movement
	 atomicAdd(eng1, e1[0]);  // energy before the particle's movement
   }
 }
 
 /**
	GPU (device code) vector binary reduction.
	@arguments:
	nitems: number of vector elements
	g_idata: vector to reduce
	@return:
	g_odata: result to reduce the input vector (sum of all vector's elements)
 */
 __global__ void __binaryReduction__(int nitems, float *g_idata,
									 float *g_odata) {
   __shared__ float sdata[NTHREAD];
   unsigned int tid = threadIdx.x;  // global thread index
   unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;  // block thread index
   if (i < nitems) {           // if global thread index is valid
	 sdata[tid] = g_idata[i];  // read data from global memory
   } else {
	 sdata[tid] = 0.0;
   }
   __syncthreads();  // sync threads
   for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
	 // for every thread in the lower middle half of the block
	 if (tid < s) {
	   // sum elements of its block index and its block index plus half the size
	   // of the block
	   sdata[tid] += sdata[tid + s];
	 }
	 __syncthreads();  // sync threads
   }
   // if block thread index is zero (the first thread of the block) atomic
   // (transactional) sum block energy to energy accumulator
   if (tid == 0) {
	 atomicAdd(g_odata, sdata[0]);
   }
 }
 
 // GPU configuration energy CUDA kernel
 __global__ void eKernel(float *r, float *eng, int natoms, int nsp, int *itp,
						 float *runit, float *rc2, float *al, float *bl,
						 float *bl2, float *cl, int *nspps, int *keyp) {
   __shared__ float rx[NTHREAD], ry[NTHREAD], rz[NTHREAD];
   float xi, yi, zi, rd2;
   int i = threadIdx.x + blockIdx.x * blockDim.x;  // global thread index
   int tx = threadIdx.x;                           // block thread index
   int imol = i * NDIM;          // particle global memory vector position
   int mtx = tx * NDIM;          // block memory vector adjust
   int rest = natoms % NTHREAD;  // particles out of block fit
   int nbl = natoms / NTHREAD;   // particles inside block fit
   int ndmol = natoms * NDIM;    // total size of global memory vector position
   double energ = 0.0;
   float rdd[NDIM];
   int iti, itj, nit;
   if (i < natoms) {  // if particle (thread) is valid
	 xi = r[imol];    // get its position from global memory
	 yi = r[imol + 1];
	 zi = r[imol + 2];
   }
   for (int m = 0; m <= nbl; ++m) {  // for every block
	 int ml = m / nbl;
	 // calculate the limit in the block fit (particle's rest is considered)
	 int lim = (1 - ml) * NTHREAD + ml * rest;
	 int mth = m * NTHREAD;       // block fit particle displacement
	 int mtt = mth * NDIM + mtx;  // block fit particle position
 
	 if (mtt <= ndmol) {  // if particle is valid
	   rx[tx] = r[mtt];   // get its position inside current block
	   ry[tx] = r[mtt + 1];
	   rz[tx] = r[mtt + 2];
	 }
	 // sync block threads (all threads/particles stores its position in current
	 // block's shared memory)
	 __syncthreads();
	 // if particle (thread) is valid
	 if (i < natoms) {
	   // for every other particle inside the block fit (check limit!)
	   for (int j = 0; j < lim; ++j) {
		 // particle global index
		 int jmth = j + mth;
		 // if particle (thread) is different of particle which calculate
		 // interaction energy
		 if (i != jmth) {
		   // get interaction potential ids for the particles
		   iti = __getIatype__(nsp, nspps, i);
		   itj = __getIatype__(nsp, nspps, jmth);
		   // get interaction potential id between particles
		   nit = itp[iti * nsp + itj];
		   // calculate distante between particle
		   rdd[0] = xi - rx[j];
		   rdd[1] = yi - ry[j];
		   rdd[2] = zi - rz[j];
		   // distance power 2 between particles
		   rd2 = __dist2__(rdd, runit);
		   // if distance is less than interaction potential cutoff radio
		   if (rd2 < rc2[nit]) {
			 // accumulate energy between particles
			 energ += __fpot__(rd2, nit, keyp, al, bl, cl, bl2);
		   }
		 }
	   }
	 }
	 __syncthreads();  // sync block threads
	 if (i < natoms) {  // if particle (thread) is valid
	   eng[i] = energ;  // set total energy to output vector
	 }
   }
 }
 
 // Set of GPU functions
 extern "C" void gpu(const int mode, const int natoms, int **itp, double *r,
					 double *runit, double *rc2, const int nsp, int *nspps,
					 int *keyp, double *al, double *bl, double *cl, double *bl2,
					 const int nitmax, const int cudadevice,
					 VSLStreamStatePtr *stream, double *rdmax, const double kt,
					 double *esr, unsigned long int *naccept, const int chpotit,
					 double *v0, double *side, double *a, double *b, double *c,
					 const double vdmax, const char *scaling, const double pres,
					 unsigned long int *nvaccept) {
   // data in GPU memory has to be static!
   static int *itpdev, *nsppsdev, *keypdev;
   static float *rdev, *runitdev, *rc2dev, *aldev, *bldev, *bl2dev, *cldev;
   if (mode == 0) {  // Initialize GPU memory.
	 hipSetDevice(cudadevice);  // set cuda device
	 // double precision to single precision temporal variables
	 float *rf = (float *)malloc(natoms * NDIM * sizeof(float));
	 int *itp_serialized = (int *)malloc(nsp * nsp * sizeof(int));
	 float *runitf = (float *)malloc(NDIM * sizeof(float));
	 float *rc2f = (float *)malloc(nitmax * sizeof(float));
	 float *alf = (float *)malloc(nitmax * sizeof(float));
	 float *blf = (float *)malloc(nitmax * sizeof(float));
	 float *bl2f = (float *)malloc(nitmax * sizeof(float));
	 float *clf = (float *)malloc(nitmax * sizeof(float));
	 if (rf == NULL || itp_serialized == NULL || runitf == NULL ||
		 rc2f == NULL || alf == NULL || blf == NULL || bl2f == NULL ||
		 clf == NULL) {
	   fputs(errorNEM, stderr);
	   exit(1);
	 }
	 for (int i = 0; i < natoms * NDIM; ++i) {
	   // convert particles positions from double precision to single precision
	   rf[i] = (float)r[i];
	 }
	 // allocate (in GPU) memory and copy particles positions from CPU to GPU
	 hipMalloc((void **)&rdev, natoms * NDIM * sizeof(float));
	 hipMemcpy(rdev, rf, natoms * NDIM * sizeof(float), hipMemcpyHostToDevice);
	 for (int i = 0; i < nsp; ++i) {
	   for (int j = 0; j < nsp; ++j) {
		 // interaction potentials between species must be serialized to take
		 // advantage of memory coalescence
		 itp_serialized[i * nsp + j] = itp[i][j];
	   }
	 }
	 // allocate (in GPU) interaction potentials between species from CPU to GPU
	 hipMalloc((void **)&itpdev, nsp * nsp * sizeof(int));
	 hipMemcpy(itpdev, itp_serialized, nsp * nsp * sizeof(int),
				hipMemcpyHostToDevice);
	 // allocate (in GPU) accumulated number of species per specie from CPU to
	 // GPU
	 hipMalloc((void **)&nsppsdev, nsp * sizeof(int));
	 hipMemcpy(nsppsdev, nspps, nsp * sizeof(int), hipMemcpyHostToDevice);
	 for (int i = 0; i < NDIM; ++i) {
	   // convert normalization units of the simulation box's length sides from
	   // double precision to single precision
	   runitf[i] = (float)runit[i];
	 }
	 // allocate (in GPU) normalization units of the simulation box's length
	 // sides from CPU to GPU
	 hipMalloc((void **)&runitdev, NDIM * sizeof(float));
	 hipMemcpy(runitdev, runitf, NDIM * sizeof(float), hipMemcpyHostToDevice);
	 // convert Morse/LJ parameters from double precision to single precision
	 for (int i = 0; i < nitmax; ++i) {
	   rc2f[i] = (float)rc2[i];
	   alf[i] = (float)al[i];
	   blf[i] = (float)bl[i];
	   bl2f[i] = (float)bl2[i];
	   clf[i] = (float)cl[i];
	 }
	 // allocate (in GPU) Morse/LJ parameters from CPU to GPU
	 hipMalloc((void **)&rc2dev, nitmax * sizeof(float));
	 hipMemcpy(rc2dev, rc2f, nitmax * sizeof(float), hipMemcpyHostToDevice);
	 hipMalloc((void **)&aldev, nitmax * sizeof(float));
	 hipMemcpy(aldev, alf, nitmax * sizeof(float), hipMemcpyHostToDevice);
	 hipMalloc((void **)&bldev, nitmax * sizeof(float));
	 hipMemcpy(bldev, blf, nitmax * sizeof(float), hipMemcpyHostToDevice);
	 hipMalloc((void **)&bl2dev, nitmax * sizeof(float));
	 hipMemcpy(bl2dev, bl2f, nitmax * sizeof(float), hipMemcpyHostToDevice);
	 hipMalloc((void **)&cldev, nitmax * sizeof(float));
	 hipMemcpy(cldev, clf, nitmax * sizeof(float), hipMemcpyHostToDevice);
	 // allocate (in GPU) potential's key from CPU to GPU
	 hipMalloc((void **)&keypdev, nitmax * sizeof(int));
	 hipMemcpy(keypdev, keyp, nitmax * sizeof(int), hipMemcpyHostToDevice);
	 // release CPU memory
	 free(rf);
	 free(itp_serialized);
	 free(runitf);
	 free(rc2f);
	 free(alf);
	 free(blf);
	 free(bl2f);
	 free(clf);
   } else if (mode == 1) {  // Energy of a configuration in parallel mode.
	 float eng;  // final energy
	 float *edev;
	 // allocate (in GPU every thread interaction energy)
	 hipMalloc((void **)&edev, natoms * sizeof(float));
	 float *engdev;
	 // allocate (in GPU) the energy accumulator
	 hipMalloc((void **)&engdev, sizeof(float));
	 // calculate the number of blocks in GPU grid
	 int nblock = natoms / NTHREAD;
	 if (natoms % NTHREAD != 0) {
	   ++nblock;
	 }
	 // call CUDA energy kernel
	 eKernel<<<nblock, NTHREAD>>>(rdev, edev, natoms, nsp, itpdev, runitdev,
								  rc2dev, aldev, bldev, bl2dev, cldev, nsppsdev,
								  keypdev);
	 // cudaCheckError();
	 // set the energy's accumulator to zero before reduction
	 hipMemset(engdev, 0, sizeof(float));
	 // call CUDA binary reduction kernel
	 __binaryReduction__<<<nblock, NTHREAD>>>(natoms, edev, engdev);
	 // cudaCheckError();
	 // copy energy accumulator from GPU to CPU
	 hipMemcpy(&eng, engdev, sizeof(float), hipMemcpyDeviceToHost);
	 hipFree(engdev);  // release GPU memory
	 hipFree(edev);
	 // return energy
	 *esr = eng / 2;
	 // Move atoms Metropolis Montecarlo algorithm in // parallel mode.
   } else if (mode == 2) {
	 int ntest, h_size = NDIM + 1;
	 float deltae, eng0, eng1;
	 double *harvest = (double *)malloc(h_size * sizeof(double));
	 double *rp = (double *)malloc(NDIM * sizeof(double));
	 float *rpf = (float *)malloc(NDIM * sizeof(float));
	 if (harvest == NULL || rpf == NULL || rp == NULL) {
	   fputs(errorNEM, stderr);
	   exit(1);
	 }
	 float *rpdev;
	 // allocate (in GPU) memory moved particle position
	 hipMalloc((void **)&rpdev, NDIM * sizeof(float));
	 float *e0dev, *e1dev;
	 // allocate (in GPU) energy accumulators (after and before the particle's
	 // movement)
	 hipMalloc((void **)&e0dev, sizeof(float));
	 hipMalloc((void **)&e1dev, sizeof(float));
	 // calculate the number of blocks in GPU grid
	 const int nblock = (natoms + (NTHREAD - 1)) / NTHREAD;
	 for (int i = 0; i < natoms; ++i) {  // for every atom/particle
	   //(*ntrial)++; // update trial counter
	   // generate uniform random numbers
	   vdRngUniform(VSL_RNG_METHOD_UNIFORM_STD, *stream, h_size, harvest, 0, 1);
	   ntest = (int)natoms * harvest[NDIM];  // set atom/particle to move
	   for (int j = 0; j < NDIM; ++j) {
		 // calculate new position for the particle to test
		 rp[j] =
			 r[ntest * NDIM + j] + rdmax[j] * (2 * harvest[j] - 1) / runit[j];
		 // check simulation box periodic conditions for particle to test
		 if (rp[j] < -0.5) {
		   rp[j] += 1;
		 }
		 if (rp[j] > 0.5) {
		   rp[j] -= 1;
		 }
		 // convert particle's positions (after the movement) from double
		 // precision to single precision
		 rpf[j] = (float)rp[j];
	   }
	   // copy particles positions (after the movement) from CPU to GPU
	   hipMemcpy(rpdev, rpf, NDIM * sizeof(float), hipMemcpyHostToDevice);
	   // set energies accumulators to zero
	   hipMemset(e0dev, 0, sizeof(float));
	   hipMemset(e1dev, 0, sizeof(float));
	   // call CUDA difference of energies kernel
	   deKernel<<<nblock, NTHREAD>>>(ntest, rdev, e0dev, e1dev, natoms, rpdev,
									 nsp, itpdev, runitdev, rc2dev, aldev, bldev,
									 bl2dev, cldev, nsppsdev, keypdev);
	   // cudaCheckError();
	   // copy energy accumulators (energy after and before the movement) from
	   // GPU to CPU
	   hipMemcpy(&eng0, e0dev, sizeof(float), hipMemcpyDeviceToHost);
	   hipMemcpy(&eng1, e1dev, sizeof(float), hipMemcpyDeviceToHost);
	   // difference of energies (after and before movement)
	   deltae = eng1 - eng0;
	   if (deltae < 0.0) {  // if diference is less than zero
		 for (int k = 0; k < NDIM; ++k) {
		   r[ntest * NDIM + k] = rp[k];  // accept the movement
		 }
		 // accept the movement inside GPU (avoid recurrent and duplicated
		 // particle positions copy between CPU and GPU)
		 // only copy the position of the moved particle
		 hipMemcpy(rdev + ntest * NDIM, rpf, NDIM * sizeof(float),
					hipMemcpyHostToDevice);
		 // update current iteration/step energy
		 *esr += deltae / (double)natoms;
		 // update number of success movements of particles in move atoms
		 // algorithm
		 (*naccept)++;
	   } else {  // else, movement not accepted by energy's difference
		 double xi[1];
		 // generate uniform random number
		 vdRngUniform(VSL_RNG_METHOD_UNIFORM_STD, *stream, 1, xi, 0, 1);
		 // if random number is less than Boltzmann entropy
		 if (exp(-deltae / kt) > xi[0]) {
		   for (int k = 0; k < NDIM; ++k) {
			 r[ntest * NDIM + k] = rp[k];  // accept the movement
		   }
		   // accept the movement inside GPU (avoid recurrent and duplicated
		   // particle positions copy between CPU and GPU)
		   // only copy the position of the moved particle
		   hipMemcpy(rdev + ntest * NDIM, rpf, NDIM * sizeof(float),
					  hipMemcpyHostToDevice);
		   // update current iteration/step energy
		   *esr += deltae / (double)natoms;
		   // update number of success movements of particles in move atoms
		   // algorithm
		   (*naccept)++;
		 }
	   }
	 }
	 // release CPU and GPU memory
	 hipFree(e0dev);
	 hipFree(e1dev);
	 hipFree(rpdev);
	 free(harvest);
	 free(rp);
	 free(rpf);
   } else if (mode == 3) {  // Chemical potental algorithm in parallel mode.
	 float chpot;           // final chemical potential
	 float *edev;
	 // GPU accumulator
	 hipMalloc((void **)&edev, sizeof(float));
	 // calculate the number of blocks in GPU grid
	 const int nblock = (chpotit + (NTHREAD - 1)) / NTHREAD;
	 const char *filename = "../results/chpotential.dat";  // output filename
	 FILE *fp;
	 // open output file in append mode
	 fp = fopen(filename, "a");
	 // random number generator deterministic mode
	 static int counter = 1;
	 // for every specie
	 for (int i = 0; i < nsp; ++i) {
	   // set excess chemical potential to zero
	   hipMemset(edev, 0, sizeof(float));
	   // chpotKernel<<<nblock, NTHREAD>>>(rdev, edev, natoms, nsp, itpdev,
	   // runitdev, rc2dev, aldev, bldev, bl2dev, cldev, nsppsdev, keypdev, i,
	   // time(NULL), chpotit, (float)kt); // random mode
	   chpotKernel<<<nblock, NTHREAD>>>(
		   rdev, edev, natoms, nsp, itpdev, runitdev, rc2dev, aldev, bldev,
		   bl2dev, cldev, nsppsdev, keypdev, i, 1 + counter, chpotit,
		   (float)kt);  // deterministic mode
	   // cudaCheckError();
	   counter++;  // random number generator deterministic mode
	   // copy excess chemical potential accumulator from GPU to CPU
	   hipMemcpy(&chpot, edev, sizeof(float), hipMemcpyDeviceToHost);
	   chpot /= chpotit;  // get average of excess chemical potential
	   // write excess chemical potential to output file
	   if (fprintf(fp, "%f\t", chpot) == EOF) {
		 printf("ERROR: cannot write to '%s' file!\n", filename);
		 exit(1);
	   }
	 }
	 if (fputc('\n', fp) == EOF) {
	   printf("ERROR: cannot write to '%s' file!\n", filename);
	   exit(1);
	 }
	 // release GPU memory and close output file
	 hipFree(edev);
	 fclose(fp);
   } else if (mode == 4) {  // Move volume algorithm in serial mode.
	 double esrOld = *esr;
	 double v0Old = *v0;
	 double *sideOld = (double *)malloc(NDIM * sizeof(double));
	 double *aOld = (double *)malloc(NDIM * sizeof(double));
	 double *bOld = (double *)malloc(NDIM * sizeof(double));
	 double *cOld = (double *)malloc(NDIM * sizeof(double));
	 double *runitOld = (double *)malloc(NDIM * sizeof(double));
	 float *runitf = (float *)malloc(NDIM * sizeof(float));
	 if (sideOld == NULL || aOld == NULL || bOld == NULL || cOld == NULL ||
		 runitOld == NULL || runitf == NULL) {
	   fputs(errorNEM, stderr);
	   exit(1);
	 }
	 // save variables before the volume movement
	 for (int i = 0; i < NDIM; ++i) {
	   sideOld[i] = side[i];
	   aOld[i] = a[i];
	   bOld[i] = b[i];
	   cOld[i] = c[i];
	   runitOld[i] = runit[i];
	 }
	 double xi[1];
	 // generate uniform random number
	 vdRngUniform(VSL_RNG_METHOD_UNIFORM_STD, *stream, 1, xi, 0, 1);
	 side[0] += (2 * xi[0] - 1) * vdmax;   // move X length side
	 if (strcmp(scaling, "ortho") == 0) {  // is scaling is ortho
	   double yzi[2];
	   // generate uniform random number
	   vdRngUniform(VSL_RNG_METHOD_UNIFORM_STD, *stream, 2, yzi, 0, 1);
	   // move Y length side using random number
	   side[1] += (2 * yzi[0] - 1) * vdmax;
	   // move Z length side using random number
	   side[2] += (2 * yzi[1] - 1) * vdmax;
	 } else {                                 // is scaling is not ortho
	   double factor = side[0] / sideOld[0];  // calculate factor side
	   side[1] *= factor;                     // move Y length side using factor
	   side[2] *= factor;                     // move Z length side using factor
	 }
	 *v0 = side[0] * side[1] * side[2];  // calculate new simulation box volume
	 // XYZ side length of the simulation box
	 a[0] = side[0];
	 b[1] = side[1];
	 c[2] = side[2];
	 for (int i = 0; i < NDIM; ++i) {
	   // set new normalization units of the simulation box's length sides
	   runit[i] = side[i];
	 }
	 for (int i = 0; i < NDIM; ++i) {
	   runitf[i] = (float)
		   runit[i];  // convert normalization units of the simulation box's
					  // length sides from double precision to single precision
	 }
	 // allocate (in GPU) normalization units of the simulation box's length
	 // sides from CPU to GPU
	 hipMemcpy(runitdev, runitf, NDIM * sizeof(float), hipMemcpyHostToDevice);
	 float eng;
	 float *edev;
	 // allocate (in GPU) the particle's interaction energy vector
	 hipMalloc((void **)&edev, natoms * sizeof(float));
	 float *engdev;
	 // allocate (in GPU) the energy accumulator
	 hipMalloc((void **)&engdev, sizeof(float));
	 // calculate the number of blocks in GPU grid
	 int nblock = natoms / NTHREAD;
	 if (natoms % NTHREAD != 0) {
	   ++nblock;
	 }
	 // call CUDA energy kernel
	 eKernel<<<nblock, NTHREAD>>>(rdev, edev, natoms, nsp, itpdev, runitdev,
								  rc2dev, aldev, bldev, bl2dev, cldev, nsppsdev,
								  keypdev);
	 // cudaCheckError();
	 // set the energy's accumulator to zero
	 hipMemset(engdev, 0, sizeof(float));
	 // call CUDA binary reduction kernel
	 __binaryReduction__<<<nblock, NTHREAD>>>(natoms, edev, engdev);
	 // cudaCheckError();
	 // copy energy accumulator from GPU to CPU
	 hipMemcpy(&eng, engdev, sizeof(float), hipMemcpyDeviceToHost);
	 // rrelease GPU memory
	 hipFree(engdev);
	 hipFree(edev);
	 // return energy
	 *esr = eng / (2 * (double)natoms);
	 // get difference of energies (after and before volume's movement)
	 double deltaEsr = *esr - esrOld;
	 // acceptance criteria for volume changes
	 double cond = exp(-(deltaEsr + pres * (*v0 - v0Old)) / kt +
					   (double)natoms * log(*v0 / v0Old));
	 // generate uniform random number
	 vdRngUniform(VSL_RNG_METHOD_UNIFORM_STD, *stream, 1, xi, 0, 1);
	 if (cond > xi[0]) {  // if random number is less than acceptance criteria
						  // for volume changes
	   (*nvaccept)++;     // accept volume movement
	 } else {             // if not accept volume movement
	   // restore variables to original values (before volume movement)
	   *esr = esrOld;
	   *v0 = v0Old;
	   for (int i = 0; i < NDIM; ++i) {
		 side[i] = sideOld[i];
		 a[i] = aOld[i];
		 b[i] = bOld[i];
		 c[i] = cOld[i];
		 runit[i] = runitOld[i];
	   }
	   for (int i = 0; i < NDIM; ++i) {
		 // convert normalization units of the simulation box's length sides from
		 // double precision to single precision
		 runitf[i] = (float)runitOld[i];
	   }
	   // allocate (in GPU) normalization units of the simulation box's length
	   // sides from CPU to GPU
	   hipMemcpy(runitdev, runitf, NDIM * sizeof(float),
				  hipMemcpyHostToDevice);
	 }
	 // release CPU memory
	 free(sideOld);
	 free(aOld);
	 free(bOld);
	 free(cOld);
	 free(runitOld);
	 free(runitf);
   } else if (mode == 5) {  // Release GPU memory
	 hipFree(itpdev);
	 hipFree(nsppsdev);
	 hipFree(keypdev);
	 hipFree(rdev);
	 hipFree(runitdev);
	 hipFree(rc2dev);
	 hipFree(aldev);
	 hipFree(bldev);
	 hipFree(bl2dev);
	 hipFree(cldev);
   } else {
	 fputs("ERROR: Incorrect GPU code!\n", stderr);
	 exit(1);
   }
 }
 